
#include <hip/hip_runtime.h>
#include<cstdlib>
#include<cassert>
#include<iostream>
#include<algorithm>
#include<functional>
#include<vector>

using std::cout;
using std::generate;
using std::vector;



__global__ void matrixMultiplication(const int* a, const int* b, int* c, int N){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    c[row * N + col] = 0;
    for(int k = 0; k < N; k++){
        c[row * N + col] += a[row * N + k] * b[k * N + col];
    }
}

void verify_result(vector<int>& a, vector<int>& b, vector<int>& c, int N){
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            int tmp = 0;
            for(int k = 0; k < N; k++){
                tmp += a[i*N + k] * b[k*N + j];
            }
            assert(tmp == c[i*N + j]);
        }
    }
}

int main(){
    const int N = 1<<10;
    size_t bytes = sizeof(int) * N * N;

    //Host
    vector<int> h_a(N*N);
    vector<int> h_b(N*N);
    vector<int> h_c(N*N);

    generate(h_a.begin(), h_a.end(), [](){return rand() % 100;});
    generate(h_b.begin(), h_b.end(), [](){return rand() % 100;});

    //Device
    int *d_a, *d_b, *d_c;
    
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

    int THREADS = 32;
    int BLOCKS = N / THREADS;

    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);

    matrixMultiplication<<<blocks, threads>>>(d_a, d_b, d_c, N);

    //
    hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);

    //
    verify_result(h_a, h_b, h_c, N);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    cout<<"Completed Successfully!\n";

    return 0;        
}
