#include "hip/hip_runtime.h"
#include<stdio.h>
#include<cassert>
#include<iostream>

__global__ void vectorAdd(int* a, int* b, int* c, int N){
    int tid = (BlockDims.x * BlockIdx.x) + ThreadIdx.x

    if(tid < N) c[tid] = a[tid] + b[tid];
}

int main(){
    const int N = 1<<16;
    size_t bytes = sizeof(int) * N;

    int *a, *b, *c;

    hipMallocManaged(&a, bytes);
    hipMallocManaged(&b, bytes);
    hipMallocManaged(&c, bytes);

    int id = hipGetDevice(&id);

    hipMemAdvise(a, bytes, cudaMemAdviseSetPreferredLoction, hipCpuDeviceId);
    hipMemAdvise(b, bytes, cudaMemAdviseSetPreferredLoction, hipCpuDeviceId);
    cudaMemPrefetchAsyc(c, bytes, id);

    for(int i = 0; i < N; i++){
        a[i] = rand() % 100;
        b[i] = rand() % 100;
    }

    hipMemAdvise(a, bytes, hipMemAdviseSetReadMostly, id);
    hipMemAdvise(b, bytes, hipMemAdviseSetReadMostly, id);
    hipMemPrefetchAsync(a, bytes, id);
    hipMemPrefetchAsync(b, bytes, id);

    int BLOCK_SIZE = 1<<10;

    int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    vectorAdd<<<GRID_SIZE, BLOCK_SIZE>>>(a, b, c, N);

    hipDeviceSynchronize();

    hipMemPrefetchAsync(a, bytes, hipCpuDeviceId);
    hipMemPrefetchAsync(b, bytes, hipCpuDeviceId);
    hipMemPrefetchAsync(c, bytes, hipCpuDeviceId);

    for(int i = 0; i < N; i++){
        assert(c[i] = a[i] + b[i]);
    }

    hipFree(a);
    hipFree(b);
    hipFree(c);

    std::cout<<"Completed Successfully!\n";

    return 0;
}